#include <iostream>
#include <hip/hip_runtime.h>

#define N 512  // number of elements

/*   
    CONFIURE LAUNCH PARAMETERS
Block and Grid Dimensions
dim3 grid_size(x,y,z);
dim3 block_size(x,y,z);

dom3 is cuda data structure. Default val (1,1,1)


    LAUNCH_KERNEL
kernelName<<< grid_size, block_size >>>(...)

cudaMalloc( location, size);
cudaFree();

cudaMemcpy( dst, src, numBytes, direction);
numBytes = N * sizeof(type)
direction
    cudaMemcpyHostToDevice
    cudaMemcpyDeviceToHost
*/



// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    } 
}

// KERNEL DEFINITION
/*  __global__ is a "declaration Specifier" that alerts the compiler 
that a function should be compiled to run in device 

KERNELS must be return typed void
    => Variable operated on in the kernel need to be passed by reference 

    We simulate "pass by reference":- Pass the address of the parameter to the kernel.


*/
__global__ void kernel(int *d_out, int *d_in ) 
{
    // Perform this operation for every thread
    d_out[0] = d_in[0];
}
/*  THREAD INDEX
Each thread has it own thread index
    Accessible within a kernel through the build in threadIdx variable

Thread Block can have as many as 3-dimension
Index of a Thread
    threadIdx.x
    threadIdx.y`
    threadIdx.z

Index of a Block
    dim3 blockIdx;
    int blockIdx.x;
    int blockIdx.y;
    int blockIdx.z;

Dimension of a Grid
    dim3 gridDim;
    int gridDim.x;    
    int gridDim.y;    
    int gridDim.z;
    
Dimension of a Block
    dim3 blockDim;
    int blockDim.x;
    int blockDim.y;
    int blockDim.z;


    INDEXING WITHIN GRID

    *   threadIdx IS ONLY Unique within its own Thread Block
    
    i = threadIdx.x + blockInd.x * blockDim.x;


    Keyword     ExexuteOn      CallOn
    ------------------------------
    __device__  Device         Device
    __global__  Device         Host
    __host__    Host           Host


    Type    Variable    Description
    -------------------------------
    dim3    gridDim     Dimensions of grid
    uint3   blockIdx    Index of block within grid
    dim3    blockDim    Dimensions of block
    unit3   threadIdx   Index of thread within block
*/

// PARALLELIZE FOR LOOP
__global__ void increment_gpu(int *a, int n) {
    int i = threadIdx.x;
    if (i < N)  //Ensure that Kernel doesnot execute more Thread than the length of the array
        a[i] = a[i] + 1;
}

void T_increment_gpu() {
    int h_a[N];

    int *d_a;
    hipMalloc( (void**)&d_a, N * sizeof(int) );

    hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice );

    dim3 grid_size(1);
    dim3 block_size(N);

    increment_gpu<<<grid_size, block_size>>>( d_a, N );
}

int main()
{
    /*  Declare variables 
    Variable that live on the host      h_
    Variable that live on the device    d_
    */ 
    int *h_c, *d_c;

    // Allocate memory on the devise
    hipMalloc( (void**)&d_c, sizeof(int) );

    // Assume h_c all ready initialize with data

    hipMemcpy(d_c, h_c, sizeof(int), hipMemcpyHostToDevice);

    // Configuration Parameters
    dim3 grid_size(1);   // Grid dimension 1 X 1 X 1 = 1 Block
    dim3 block_size(1);  // Blpck Dimension 1 X 1 X 1 = 1 Thread

    // kernel<<< grid_size, block_size >>>(...);


    hipMemcpy(h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    // De-allocatememory
    hipFree( d_c );
    free( h_c );


    // Host vectors
    float h_A[N], h_B[N], h_C[N];

    // Initialize host data
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Device vectors
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel with 1 block of N threads
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost );

    // Verify results
    for (int i = 0; i < 10; i++) {
        std::cout << "h_C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
